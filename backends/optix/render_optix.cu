#include "hip/hip_runtime.h"
#include "cuda_utils.h"
#include "lcg_rng.h"
#include "disney_bsdf.h"
#include "lights.h"
#include "optix_params.h"
#include "util/texture_channel_mask.h"

extern "C" {
    __constant__ LaunchParams launch_params;
}

struct RayPayload {
    // payload registers 0, 1
    float2 uv;
    // payload register 2
    float t_hit;
    // payload register 3
    uint32_t material_id;
    // payload registers 4, 5, 6
    float3 normal;
};

__device__ RayPayload make_ray_payload() {
    RayPayload p;
    p.uv = make_float2(0.f);
    p.t_hit = -1.f;
    p.material_id = 0;
    p.normal = make_float3(0.f);
    return p;
}

__device__ float textured_scalar_param(const float x, const float2 &uv) {
    const uint32_t mask = __float_as_int(x);
    if (IS_TEXTURED_PARAM(mask)) {
        const uint32_t tex_id = GET_TEXTURE_ID(mask);
        const uint32_t channel = GET_TEXTURE_CHANNEL(mask);
        return component(tex2D<float4>(launch_params.textures[tex_id], uv.x, uv.y), channel);
    }
    return x;
}

__device__ void unpack_material(const MaterialParams &p, float2 uv, DisneyMaterial &mat) {
    uint32_t mask = __float_as_int(p.base_color.x);
    if (IS_TEXTURED_PARAM(mask)) {
        const uint32_t tex_id = GET_TEXTURE_ID(mask);
        mat.base_color = make_float3(tex2D<float4>(launch_params.textures[tex_id], uv.x, uv.y));
    } else {
        mat.base_color = p.base_color;
    }

    mat.metallic = textured_scalar_param(p.metallic, uv);
    mat.specular = textured_scalar_param(p.specular, uv);
    mat.roughness = textured_scalar_param(p.roughness, uv);
    mat.specular_tint = textured_scalar_param(p.specular_tint, uv);
    mat.anisotropy = textured_scalar_param(p.anisotropy, uv);
    mat.sheen = textured_scalar_param(p.sheen, uv);
    mat.sheen_tint = textured_scalar_param(p.sheen_tint, uv);
    mat.clearcoat = textured_scalar_param(p.clearcoat, uv);
    mat.clearcoat_gloss = textured_scalar_param(p.clearcoat_gloss, uv);
    mat.ior = textured_scalar_param(p.ior, uv);
    mat.specular_transmission = textured_scalar_param(p.specular_transmission, uv);
}

__device__ float3 sample_direct_light(const DisneyMaterial &mat, const float3 &hit_p,
        const float3 &n, const float3 &v_x, const float3 &v_y, const float3 &w_o,
        const QuadLight *lights, const uint32_t num_lights, uint16_t &ray_count, LCGRand &rng)
{
    float3 illum = make_float3(0.f);

    uint32_t light_id = lcg_randomf(rng) * num_lights;
    light_id = min(light_id, num_lights - 1);
    QuadLight light = lights[light_id];

    const uint32_t occlusion_flags = OPTIX_RAY_FLAG_DISABLE_ANYHIT
        | OPTIX_RAY_FLAG_TERMINATE_ON_FIRST_HIT
        | OPTIX_RAY_FLAG_DISABLE_CLOSESTHIT;

    // Sample the light to compute an incident light ray to this point
    {
        float3 light_pos = sample_quad_light_position(light,
                make_float2(lcg_randomf(rng), lcg_randomf(rng)));
        float3 light_dir = light_pos - hit_p;
        float light_dist = length(light_dir);
        light_dir = normalize(light_dir);

        float light_pdf = quad_light_pdf(light, light_pos, hit_p, light_dir);
        float bsdf_pdf = disney_pdf(mat, n, w_o, light_dir, v_x, v_y);

        uint32_t shadow_hit = 1;
        optixTrace(launch_params.scene, hit_p, light_dir, EPSILON, light_dist, 0.f,
                0xff, occlusion_flags, PRIMARY_RAY, 1, OCCLUSION_RAY,
                shadow_hit);
#ifdef REPORT_RAY_STATS
        ++ray_count;
#endif
        if (light_pdf >= EPSILON && bsdf_pdf >= EPSILON && !shadow_hit) {
            float3 bsdf = disney_brdf(mat, n, w_o, light_dir, v_x, v_y);
            float w = power_heuristic(1.f, light_pdf, 1.f, bsdf_pdf);
            illum = bsdf * light.emission * fabs(dot(light_dir, n)) * w / light_pdf;
        }
    }

    // Sample the BRDF to compute a light sample as well
    {
        float3 w_i;
        float bsdf_pdf;
        float3 bsdf = sample_disney_brdf(mat, n, w_o, v_x, v_y, rng, w_i, bsdf_pdf);

        float light_dist;
        float3 light_pos;
        if (!all_zero(bsdf) && bsdf_pdf >= EPSILON && quad_intersect(light, hit_p, w_i, light_dist, light_pos)) {
            float light_pdf = quad_light_pdf(light, light_pos, hit_p, w_i);
            if (light_pdf >= EPSILON) {
                float w = power_heuristic(1.f, bsdf_pdf, 1.f, light_pdf);
                uint32_t shadow_hit = 1;
                optixTrace(launch_params.scene, hit_p, w_i, EPSILON, light_dist, 0.f,
                        0xff, occlusion_flags, PRIMARY_RAY, 1, OCCLUSION_RAY,
                        shadow_hit);
#ifdef REPORT_RAY_STATS
                ++ray_count;
#endif
                if (!shadow_hit) {
                    illum = illum + bsdf * light.emission * fabs(dot(w_i, n)) * w / bsdf_pdf;
                }
            }
        }
    }
    return illum;
}

extern "C" __global__ void __raygen__perspective_camera() {
    const RayGenParams &params = get_shader_params<RayGenParams>();

    const uint2 pixel = make_uint2(optixGetLaunchIndex().x, optixGetLaunchIndex().y);
    const uint2 screen = make_uint2(optixGetLaunchDimensions().x, optixGetLaunchDimensions().y);
    const uint32_t pixel_idx = pixel.x + pixel.y * screen.x;

    LCGRand rng = get_rng(launch_params.frame_id);
    const float2 d = make_float2(pixel.x + lcg_randomf(rng), pixel.y + lcg_randomf(rng)) / make_float2(screen);
    float3 ray_dir = normalize(d.x * make_float3(launch_params.cam_du)
            + d.y * make_float3(launch_params.cam_dv) + make_float3(launch_params.cam_dir_top_left));

    float3 ray_origin = make_float3(launch_params.cam_pos);

    DisneyMaterial mat;

    uint16_t ray_count = 0;
    const float3 light_emission = make_float3(1.f);
    int bounce = 0;
    float3 illum = make_float3(0.f);
    float3 path_throughput = make_float3(1.f);
    do {
        RayPayload payload = make_ray_payload();

        optixTrace(launch_params.scene,
                ray_origin,
                ray_dir,
                EPSILON,
                1e20f,
                0.f,
                0xff,
                OPTIX_RAY_FLAG_DISABLE_ANYHIT,
                PRIMARY_RAY,
                1,
                PRIMARY_RAY,
                reinterpret_cast<uint32_t&>(payload.uv.x),
                reinterpret_cast<uint32_t&>(payload.uv.y),
                reinterpret_cast<uint32_t&>(payload.t_hit),
                payload.material_id,
                reinterpret_cast<uint32_t&>(payload.normal.x),
                reinterpret_cast<uint32_t&>(payload.normal.y),
                reinterpret_cast<uint32_t&>(payload.normal.z));
#ifdef REPORT_RAY_STATS
        ++ray_count;
#endif

        if (payload.t_hit <= 0.f) {
            illum = illum + path_throughput * payload.normal;
            break;
        }

        unpack_material(params.materials[payload.material_id], payload.uv, mat);

        const float3 w_o = -ray_dir;
        const float3 hit_p = ray_origin + payload.t_hit * ray_dir;
        float3 v_x, v_y;
        float3 v_z = payload.normal;
        if (mat.specular_transmission == 0.f && dot(w_o, v_z) < 0.f) {
            v_z = -v_z;
        }
        ortho_basis(v_x, v_y, v_z);

        illum = illum + path_throughput * sample_direct_light(mat, hit_p, v_z, v_x, v_y, w_o,
                params.lights, params.num_lights, ray_count, rng);

        float3 w_i;
        float pdf;
        float3 bsdf = sample_disney_brdf(mat, v_z, w_o, v_x, v_y, rng, w_i, pdf);
        if (pdf == 0.f || all_zero(bsdf)) {
            break;
        }
        path_throughput = path_throughput * bsdf * fabs(dot(w_i, v_z)) / pdf;

        ray_origin = hit_p;
        ray_dir = w_i;
        ++bounce;

        // Russian roulette termination
        if (bounce > 3) {
            const float q = max(0.05f, 1.f - max(path_throughput.x, max(path_throughput.y, path_throughput.z)));
            if (lcg_randomf(rng) < q) {
                break;
            }
            path_throughput = path_throughput / (1.f - q);
        }
    } while (bounce < MAX_PATH_DEPTH);

    const float3 prev_color = make_float3(launch_params.accum_buffer[pixel_idx]);
    const float3 accum_color = (illum + launch_params.frame_id * prev_color) / (launch_params.frame_id + 1);
    launch_params.accum_buffer[pixel_idx] = make_float4(accum_color, 1.f);

    launch_params.framebuffer[pixel_idx] = make_uchar4(
            clamp(linear_to_srgb(accum_color.x) * 255.f, 0.f, 255.f),
            clamp(linear_to_srgb(accum_color.y) * 255.f, 0.f, 255.f),
            clamp(linear_to_srgb(accum_color.z) * 255.f, 0.f, 255.f), 255);

#ifdef REPORT_RAY_STATS
    launch_params.ray_stats_buffer[pixel_idx] = ray_count;
#endif
}

extern "C" __global__ void __miss__miss() {
    optixSetPayload_1(float_as_int(-1.f));
    float3 dir = optixGetWorldRayDirection();
    // Apply our miss "shader" to draw the checkerboard background
    float u = (1.f + atan2(dir.x, -dir.z) * M_1_PI) * 0.5f;
    float v = acos(dir.y) * M_1_PI;

    int check_x = u * 10.f;
    int check_y = v * 10.f;

    if (dir.y > -0.1f && (check_x + check_y) % 2 == 0) {
        optixSetPayload_4(float_as_int(0.5f));
        optixSetPayload_5(float_as_int(0.5f));
        optixSetPayload_6(float_as_int(0.5f));
    } else {
        optixSetPayload_4(float_as_int(0.1f));
        optixSetPayload_5(float_as_int(0.1f));
        optixSetPayload_6(float_as_int(0.1f));
    }
}

extern "C" __global__ void __miss__occlusion_miss() {
    optixSetPayload_0(0);
}

extern "C" __global__ void __closesthit__closest_hit() {
    const HitGroupParams &params = get_shader_params<HitGroupParams>();

    const float2 bary = optixGetTriangleBarycentrics();
    const uint3 indices = params.index_buffer[optixGetPrimitiveIndex()];
    const float3 v0 = params.vertex_buffer[indices.x];
    const float3 v1 = params.vertex_buffer[indices.y];
    const float3 v2 = params.vertex_buffer[indices.z];
    float3 normal = normalize(cross(v1 - v0, v2 - v0));
    normal = normalize(optixTransformNormalFromObjectToWorldSpace(normal));

    float2 uv = make_float2(0.f);
    if (params.uv_buffer) {
        float2 uva = params.uv_buffer[indices.x];
        float2 uvb = params.uv_buffer[indices.y];
        float2 uvc = params.uv_buffer[indices.z];
        uv = (1.f - bary.x - bary.y) * uva
            + bary.x * uvb + bary.y * uvc;
    }

    optixSetPayload_0(float_as_int(uv.x));
    optixSetPayload_1(float_as_int(uv.y));

    optixSetPayload_2(float_as_int(optixGetRayTmax()));
    optixSetPayload_3(params.material_id);

    optixSetPayload_4(float_as_int(normal.x));
    optixSetPayload_5(float_as_int(normal.y));
    optixSetPayload_6(float_as_int(normal.z));
}

